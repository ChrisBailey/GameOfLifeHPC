/**
 * @file gol-cuda.cu
 *
 * @brief A finite 2-D NxM toroidal implementation of Conway's Game of Life using CUDA.
 *
 * @author Christopher Bailey
 * @date 2022
 * 
 * THIS PROGRAM IS WRITTEN USING CUDA AND REQUIRES
 * USING "module load cuda" ON GETAFIX
 * 
 * FOR CONSOLE OUTPUT UNCOMMENT THE MARKED LINES IN THE RUN METHOD
 */


#include <hip/hip_runtime.h>
#include <algorithm>
#include <chrono>
#include <fstream>
#include <functional>
#include <iostream>
#include <random>
#include <string>
#include <vector>

/***
 * CUDA error checking function used from sumarrays-gpu-v1.cu in course git
 */
void checkError(hipError_t e)
{
   if (e != hipSuccess)
   {
      std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
      abort();
   }
}

/***
 * Implementation of the neighbour count and and ruleset on a CUDA device
 * 
 * Code will run on device but can be called from the CPU
 */
__global__
void cudaRun(int n, int w, int h, int* current, int* next) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;

    for (int i = index; i < n; i+= stride) {
        const int x = i % w;
        const int y = i / w;

        // count neighbours
        const int neighbours =
            current[((x - 1 + w) % w) + (((y - 1 + h) % h) * w)] // top left
            + current[(x) + (((y - 1 + h) % h) * w)] // top
            + current[((x + 1) % w) + (((y - 1 + h) % h) * w)] // top right;
            + current[((x - 1 + w) % w) + y*w] // left
            + current[((x + 1) % w) + y*w] // right
            + current[((x - 1 + w) % w) + (((y + 1) % h) * w)] // bottom left
            + current[(x) + (((y + 1) % h) * w)] // bottom
            + current[((x + 1) % w) + (((y + 1) % h) * w)]; // bottom right;

        // ruleset implementation logic
        if (neighbours == 2 ) {  // maintain
            next[i] = current[i];
        } else if (neighbours == 3) {  // live
            next[i] = 1;
        } else {  // die
            next[i] = 0;
        }
    }
}


/***
 * Implements the Game of Life
 */
class Life {
public:
    /***
    * Constructor for initialising
    */
    Life(const unsigned h, const unsigned w) {
        height = h;
        width = w;

        cells.resize(height*width);
        nextcells.resize(height*width);

        // init device memory
        checkError(hipMalloc(&cellsDevice, cells.size()*sizeof(int)));
        checkError(hipMalloc(&nextcellsDevice, cells.size()*sizeof(int)));

        // set board pattern here
        init_random();

        // copy cells from host to device
        checkError(hipMemcpy(cellsDevice, cells.data(), cells.size()*sizeof(int), hipMemcpyHostToDevice));
    }

    /***
     * Simulates the Game of Life
     */
    void run(const unsigned numTurns) {
        // uncomment below line to print to console
        //output_print();

        // we want to use the max number of CUDA threads per CUDA thread block for performance
        // we want to make sure we can use as many blocks as the problem and device will fit
        // this is device dependent on number of cuda cores and cuda streaming multiprocessors
        const int threads = 256;
        const int blocks = (cells.size()+threads-1)/threads;

        for (unsigned t = 0; t < numTurns; t++) {
            // run the computation on the device
            cudaRun<<<blocks, threads>>>(cells.size(), width, height, cellsDevice, nextcellsDevice); // asychronous
            checkError(hipDeviceSynchronize());
            
            // swap between the grids so we dont have to do expensive copies
            int* temp = cellsDevice;
            cellsDevice = nextcellsDevice;
            nextcellsDevice = temp;

            // copy nextcells from device to host
            // this is only necessary for console output and can be moved outside the loop to save io time
            checkError(hipMemcpy(cells.data(), cellsDevice, cells.size()*sizeof(int), hipMemcpyDeviceToHost));

            // uncomment below lines to print to console
            //std::cout << "t:" << t << std::endl;
            //output_print();
        }
    }

    /***
     * Writes the current board state to a file called output.txt
     */
    void output_file() const {
        std::ofstream outFile("output-cuda.txt");
        for (unsigned y = 0; y < height; y++) {
            for (unsigned x = 0; x < width; x++) {
                outFile << (unsigned)cells[y*width + x];
            }
            outFile << std::endl;
        }
        outFile.close();
    }

    /***
     * Writes the current board state to the console
     */
    void output_print() const {
        std::cout << std::string(width, '-') << std::endl;
        for (unsigned y = 0; y < height; y++) {
            for (unsigned x = 0; x < width; x++) {
                std::cout << (unsigned)cells[y*width + x];
            }
            std::cout << std::endl;
        }
        std::cout << std::string(width, '-') << std::endl << std::endl;
    }

private:
    // tunable model parameters
    unsigned height;
    unsigned width;

    // non-tunable model parameters
    const int alive = 1;
    const int dead = 0;
    std::vector<int> cells;
    std::vector<int> nextcells;
    int* cellsDevice;
    int* nextcellsDevice;

    /***
     * Fills the board with random dead/alive cells
     */
    void init_random() {
        std::uniform_int_distribution<int> distribution(0,1);
        std::mt19937 engine;
        auto generator = std::bind(distribution, engine);
        std::generate(cells.begin(), cells.end(), generator);
    }

    /***
     * Fills the board with a single glider in the top left corner
     */
    void init_glider() {
        std::fill(cells.begin(), cells.end(), dead);

        cells[0+2] = alive;
        cells[0+width] = alive;
        cells[0+width+2] = alive;
        cells[0+width+width+1] = alive;
        cells[0+width+width+2] = alive;
    }

    /***
     * Fills the board with a provided pattern file in x format
     */
    void init_pattern() {
        // TODO: implement pattern loading
    }
};


/***
 * Program entry point
 */
int main(int argc, char *argv[]) {
    auto totalStartTime = std::chrono::steady_clock::now();
    // command line argument handling
    std::vector<std::string> args(argv, argv+argc);
    unsigned height, width, turns;
    if (args.size() == 4) {
        height = std::stoi(args[1]);
        width = std::stoi(args[2]);
        turns = std::stoi(args[3]);
    } else {
        std::cout << "Usage: ./gol-cuda <height> <width> <turns>" << std::endl;
        return 1;
    }

    Life l(height, width);

    // time and run the simulation
    auto modelStartTime = std::chrono::steady_clock::now();
    l.run(turns);
    auto modelFinishTime = std::chrono::steady_clock::now();
    auto modelTime = std::chrono::duration_cast<std::chrono::microseconds>(modelFinishTime - modelStartTime);
    std::cout << "Model run time: " << modelTime.count() << " us\n";

    // output and cleanup
    l.output_file();
    auto totalFinishTime = std::chrono::steady_clock::now();
    auto totalTime = std::chrono::duration_cast<std::chrono::microseconds>(totalFinishTime - totalStartTime);
    std::cout << "Total time: " << totalTime.count() << " us\n";
}
